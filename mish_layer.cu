#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/neuron_layer.hpp"
#include "caffe/layers/mish_layer.hpp"

namespace caffe {

// CUDA kernele for forward
template <typename Dtype>
__global__ void MishForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
      Dtype x=in[index];
      Dtype tmp=exp(x)+1;
      out[index]=x-2*x/(tmp*tmp+1);
  }
}

// CUDA kernel for bottom backward
template <typename Dtype>
__global__ void MishBackward(const int n, const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype x=in_data[index];
    Dtype tmp=exp(x)+1;
    Dtype tmp1=tmp*tmp+1;
    Dtype diff_tmp=1-2/tmp1+4*x*(tmp*tmp-tmp)/(tmp1*tmp1);
    out_diff[index]=in_diff[index]*diff_tmp;
  }
}

template <typename Dtype>
void MishLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();  
  //Dtype* backward_buff_data=backward_buff_.mutable_gpu_data();
  // For in-place computation
  if (top[0] == bottom[0]) {
    caffe_copy(count, bottom_data, backward_buff_.mutable_gpu_data());
  }

  // NOLINT_NEXT_LINE(whitespace/operators)
  MishForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void MishLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {

  const Dtype* top_diff = top[0]->gpu_diff();
  const int count = bottom[0]->count();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  if (top[0] == bottom[0]) {
    bottom_data = backward_buff_.gpu_data();
  }
  Dtype* bottom_diff= bottom[0]->mutable_gpu_diff();

  // Propagate to bottom
  if (propagate_down[0]) {
    MishBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(MishLayer);
}  // namespace caffe
